#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matmul_tiled(float *A, float *B, float *C, int n) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;

    for (int t = 0; t < n / TILE_SIZE; ++t) {
        As[threadIdx.y][threadIdx.x] = A[row * n + t * TILE_SIZE + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * n + col];
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        __syncthreads();
    }
    C[row * n + col] = sum;
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    const int N = atoi(argv[1]);
    if (N % TILE_SIZE != 0) {
        printf("Error: Matrix size must be a multiple of %d\n", TILE_SIZE);
        return 1;
    }

    const int trials = 10;
    const size_t size = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N*N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel configuration
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid(N / TILE_SIZE, N / TILE_SIZE);

    // Timing setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_ms = 0;

    for (int i = 0; i < trials; ++i) {
        hipMemset(d_C, 0, size); // Reset output
        hipEventRecord(start, 0);
        matmul_tiled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_ms += ms;
    }

    // Copy result back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("[Tiled] Matrix %dx%d - Avg time (%d trials): %.3f ms\n", 
           N, N, trials, total_ms / trials);
    printf("Validation: C[0] = %.1f\n", h_C[0]);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
