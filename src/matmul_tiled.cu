#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matmul_tiled(float *A, float *B, float *C, int n) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;

    for (int t = 0; t < (n + TILE_SIZE -1)/TILE_SIZE; ++t) {
        int aCol = t*TILE_SIZE + threadIdx.x;
        int bRow = t*TILE_SIZE + threadIdx.y;
        As[threadIdx.y][threadIdx.x] = (row<n && aCol<n) ? A[row*n + aCol] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (bRow<n && col<n) ? B[bRow*n + col] : 0.0f;
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        __syncthreads();
    }
    if (row<n && col<n) C[row*n + col] = sum;
}

int main(int argc, char **argv) {
    if (argc!=2) { printf("Usage: %s <matrix_size>\n", argv[0]); return 1; }
    int N = atoi(argv[1]);
    const int trials=10; size_t bytes=N*N*sizeof(float);

    float *h_A=(float*)malloc(bytes), *h_B=(float*)malloc(bytes), *h_C=(float*)malloc(bytes);
    for(int i=0;i<N*N;++i){h_A[i]=1.0f; h_B[i]=2.0f;}

    float *d_A,*d_B,*d_C;
    hipMalloc(&d_A,bytes); hipMalloc(&d_B,bytes); hipMalloc(&d_C,bytes);
    hipMemcpy(d_A,h_A,bytes,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,bytes,hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE,TILE_SIZE), grid((N+TILE_SIZE-1)/TILE_SIZE,(N+TILE_SIZE-1)/TILE_SIZE);

    hipEvent_t start,stop; hipEventCreate(&start); hipEventCreate(&stop);
    float total_ms=0;
    for(int t=0;t<trials;++t){
        hipMemset(d_C,0,bytes);
        hipEventRecord(start);
        matmul_tiled<<<grid,block>>>(d_A,d_B,d_C,N);
        hipError_t err=hipGetLastError();
        if(err!=hipSuccess){printf("Kernel failed: %s\n",hipGetErrorString(err));return 1;}
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms; hipEventElapsedTime(&ms,start,stop);
        total_ms+=ms;
    }

    hipMemcpy(h_C,d_C,bytes,hipMemcpyDeviceToHost);
    printf("[Tiled]  N=%d  AvgTime=%.3f ms\n", N, total_ms/trials);
    printf("Validation C[0]=%.1f\n", h_C[0]);
    fflush(stdout);

    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
