#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024
#define TILE_SIZE 16

__global__ void matmul_tiled(float *A, float *B, float *C, int n) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;

    for (int t = 0; t < n / TILE_SIZE; ++t) {
        // Coalesced global memory access
        As[threadIdx.y][threadIdx.x] = A[row * n + t * TILE_SIZE + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * n + col];
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        __syncthreads();
    }
    C[row * n + col] = sum;
}

int main() {
    int size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for (int i = 0; i < N*N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N+TILE_SIZE-1)/TILE_SIZE, (N+TILE_SIZE-1)/TILE_SIZE);

    // --- Timing Start ---
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matmul_tiled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // --- Timing End ---

    // Calculate and print elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[Tiled] Kernel execution time: %.3f ms\n", milliseconds);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("C[0]=%f\n", C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);
    return 0;
}
