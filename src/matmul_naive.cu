#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matmul_naive(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    const int N = atoi(argv[1]);
    const int trials = 10;
    const size_t size = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N*N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel configuration
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (N + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // Timing setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_ms = 0;

    for (int i = 0; i < trials; ++i) {
        hipMemset(d_C, 0, size); // Reset output
        hipEventRecord(start, 0);
        matmul_naive<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_ms += ms;
    }

    // Copy result back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("[Naive] Matrix %dx%d - Avg time (%d trials): %.3f ms\n", 
           N, N, trials, total_ms / trials);
    printf("Validation: C[0] = %.1f\n", h_C[0]);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
