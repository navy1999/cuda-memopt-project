#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  // Matrix size N x N

__global__ void matmul_naive(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < n && col < n) {
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N*N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N+15)/16, (N+15)/16);

    // --- Timing Start ---
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matmul_naive<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // --- Timing End ---

    // Calculate and print elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[Naive] Kernel execution time: %.3f ms\n", milliseconds);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("C[0]=%f\n", C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);
    return 0;
}
